
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

using namespace std;

const int CONST = 128;

void init(uint64_t size, float *A, float *B, float *C) {
  for (size_t i = 0; i < size; i++) {
    A[i] = static_cast<float>(rand() % 100);
    B[i] = static_cast<float>(rand() % 100);
    C[i] = static_cast<float>(rand() % 100);
  }
}

__global__ void kernelA(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    if (x[i] > y[i]) {
      for (int j = 0; j < n/CONST; j++)
        y[i] = x[j] + y[j];
    }
    else {
      for (int j = 0; j < n/CONST; j++)
        y[i] = x[j] / y[j];
    }
  }
}

__global__ void kernelB(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    if (x[i] > y[i]) {
      for (int j = 0; j < n/CONST; j++)
        y[i] = x[j] + y[j];
    }
    else {
      y[i] = atomicAdd(&y[i], x[i]);
    }
  }
}

__global__ void kernelC(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    if (x[i] > y[i]) {
      for (int j = 0; j < n/CONST; j++)
        y[i] = x[j] + y[j];
    }
}

__global__ void kernelD(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    for (int j = 0; j < n/CONST; j++)
      y[i] = atomicAdd(&y[j], x[j]);
  }
}

int main(int argc, char *argv[]) {
  int size;
  if (argc == 2) {
    cout << "\nArray size: " << argv[1] << endl;
    size = atoi(argv[1]);
  } else {
    size = 1 << 16;
    cout << "\nUsing default matrix size: " << size << endl;
  }

  const int nStreams = 4;

  // One cudaGraphExec_t per stream is required
  hipGraph_t graph;
  bool graphCreated[nStreams];
  for (int i = 0; i < nStreams; i++)
    graphCreated[i] = false;
  hipGraphExec_t instance[nStreams];

  // Declare host data
  float *A_h[nStreams];
  float *B_h[nStreams];
  float *C_h[nStreams];

  for (int i = 0; i < nStreams; i++) {
    hipHostMalloc(reinterpret_cast<void **>(&A_h[i]), size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(reinterpret_cast<void **>(&B_h[i]), size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(reinterpret_cast<void **>(&C_h[i]), size * sizeof(float), hipHostMallocDefault);
  }

  // Declare device data
  float *A_d[nStreams];
  float *B_d[nStreams];
  float *C_d[nStreams];
  for (int i = 0; i < nStreams; i++) {
    hipMalloc(reinterpret_cast<void **>(&A_d[i]), size * sizeof(float));
    hipMalloc(reinterpret_cast<void **>(&B_d[i]), size * sizeof(float));
    hipMalloc(reinterpret_cast<void **>(&C_d[i]), size * sizeof(float));
  }

  // Initialize host data
  for (int i = 0; i < nStreams; i++)
    init(size, A_h[i], B_h[i], C_h[i]);

  // Create CUDA events for timing measurement
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // dim3 gridDim;
  // dim3 blockDim;
  const int gridDim = 1024;
  const int blockDim = 64;

  hipStream_t stream[nStreams];

  for (size_t i = 0; i < nStreams; i++)
    hipStreamCreate(&stream[i]);
  
  hipEventRecord(start);

  for (size_t i = 0; i < 1000; i++) {
    int idStream = i % nStreams;

    // Create graph if not exits
    if(!graphCreated[idStream]){
      hipStreamBeginCapture(stream[idStream], hipStreamCaptureModeGlobal);  // begin of the graph
      hipMemcpyAsync(reinterpret_cast<void *>(A_d[idStream]), reinterpret_cast<void *>(A_h[idStream]), size,
                 hipMemcpyHostToDevice, stream[idStream]);
      hipMemcpyAsync(reinterpret_cast<void *>(B_d[idStream]), reinterpret_cast<void *>(B_h[idStream]), size,
                 hipMemcpyHostToDevice, stream[idStream]);
      kernelA<<<gridDim, blockDim, 0, stream[idStream]>>>(size, A_d[idStream], B_d[idStream]);

      hipMemcpyAsync(reinterpret_cast<void *>(C_d[idStream]), reinterpret_cast<void *>(C_h[idStream]), size,
                 hipMemcpyHostToDevice, stream[idStream]);
      kernelB<<<gridDim, blockDim, 0, stream[idStream]>>>(size, B_d[idStream], C_d[idStream]);

      kernelC<<<gridDim, blockDim, 0, stream[idStream]>>>(size, C_d[idStream], A_d[idStream]);
      hipMemcpyAsync(reinterpret_cast<void *>(C_d[idStream]), reinterpret_cast<void *>(C_h[idStream]), size,
                 hipMemcpyHostToDevice, stream[idStream]);

      kernelD<<<gridDim, blockDim, 0, stream[idStream]>>>(size, A_d[idStream], B_d[idStream]);
      hipMemcpyAsync(reinterpret_cast<void *>(A_d[idStream]), reinterpret_cast<void *>(A_h[idStream]), size,
                 hipMemcpyHostToDevice, stream[idStream]);
      hipMemcpyAsync(reinterpret_cast<void *>(B_d[idStream]), reinterpret_cast<void *>(B_h[idStream]), size,
                 hipMemcpyHostToDevice, stream[idStream]);
      hipStreamEndCapture(stream[idStream], &graph); // end of the graph
      hipGraphInstantiate(&instance[idStream], graph, NULL, NULL, 0);
      graphCreated[idStream]=true;
    }
    // Otherwise launch graph directly
    hipGraphLaunch(instance[idStream], stream[idStream]);
  }

  hipEventRecord(stop);

  // Print total runtime
  hipEventSynchronize(stop);
  float milliseconds = 0.f;
  hipEventElapsedTime(&milliseconds, start, stop);
  double seconds = static_cast<double>(milliseconds) / 1000.;
  cout << "runtime: " << seconds << endl;

  // Print to prevent the compiler from over optimization
  for (size_t i = 0; i < nStreams; i++) {
    cout << A_h[i][CONST] << endl;
    cout << B_h[i][CONST] << endl;
    cout << C_h[i][CONST] << endl;
  }
  
  for (size_t i = 0; i < nStreams; i++)
    hipStreamDestroy(stream[i]);

  // Free the allocated memory
  for (size_t i = 0; i < nStreams; i++) {
    hipHostFree(A_h[i]);
    hipHostFree(B_h[i]);
    hipHostFree(C_h[i]);
    hipFree(reinterpret_cast<void *>(A_d[i]));
    hipFree(reinterpret_cast<void *>(B_d[i]));
    hipFree(reinterpret_cast<void *>(C_d[i]));
  }

  return 0;
}
